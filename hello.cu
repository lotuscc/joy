#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hello.h"


__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}


void say_hello(){
    cuda_hello<<<10, 100>>>();

    hipDeviceSynchronize();
}
